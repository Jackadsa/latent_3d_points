
#include <hip/hip_runtime.h>
//n<=4096, m<=1024
__global__ void approxmatch(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,float * __restrict__ match){
	const int MaxN=4096,MaxM=1024;
	__shared__ float remainL[MaxN],remainR[MaxM],ratioR[MaxM],ratioL[MaxN];
	__shared__ int listR[MaxM],lc;
	float multiL,multiR;
	if (n>=m){
		multiL=1;
		multiR=n/m;
	}else{
		multiL=m/n;
		multiR=1;
	}
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x;j<n*m;j+=blockDim.x)
			match[i*n*m+j]=0;
		for (int j=threadIdx.x;j<n;j+=blockDim.x)
			remainL[j]=multiL;
		for (int j=threadIdx.x;j<m;j+=blockDim.x)
			remainR[j]=multiR;
		__syncthreads();
		for (int j=7;j>=-2;j--){
			float level=-powf(4.0f,j);
			if (j==-2){
				level=0;
			}
			if (threadIdx.x==0){
				lc=0;
				for (int k=0;k<m;k++)
					if (remainR[k]>0)
						listR[lc++]=k;
			}
			__syncthreads();
			int _lc=lc;
			for (int k=threadIdx.x;k<n;k+=blockDim.x){
				float suml=1e-9f;
				float x1=xyz1[(i*n+k)*2+0];
				float y1=xyz1[(i*n+k)*2+1];
				//for (int l=0;l<m;l++){
				for (int _l=0;_l<_lc;_l++){
					int l=listR[_l];
					float x2=xyz2[(i*m+l)*2+0]-x1;
					float y2=xyz2[(i*m+l)*2+1]-y1;
					float w=expf(level*(x2*x2+y2*y2))*remainR[l];
					suml+=w;
				}
				ratioL[k]=remainL[k]/suml;
			}
			__syncthreads();
			//for (int k=threadIdx.x;k<m;k+=blockDim.x){
			for (int _k=threadIdx.x;_k<lc;_k+=blockDim.x){
				int k=listR[_k];
				float sumr=0;
				float x2=xyz2[(i*m+k)*2+0];
				float y2=xyz2[(i*m+k)*2+1];
				for (int l=0;l<n;l++){
					float x1=xyz1[(i*n+l)*2+0]-x2;
					float y1=xyz1[(i*n+l)*2+1]-y2;
					float w=expf(level*(x1*x1+y1*y1))*ratioL[l];
					sumr+=w;
				}
				sumr*=remainR[k];
				float consumption=fminf(remainR[k]/(sumr+1e-9f),1.0f);
				ratioR[k]=consumption*remainR[k];
				remainR[k]=fmaxf(0.0f,remainR[k]-sumr);
			}
			__syncthreads();
			for (int k=threadIdx.x;k<n;k+=blockDim.x){
				float suml=0;
				float x1=xyz1[(i*n+k)*2+0];
				float y1=xyz1[(i*n+k)*2+1];
				for (int _l=0;_l<_lc;_l++){
					int l=listR[_l];
					float x2=xyz2[(i*m+l)*2+0]-x1;
					float y2=xyz2[(i*m+l)*2+1]-y1;
					float w=expf(level*(x2*x2+y2*y2))*ratioL[k]*ratioR[l];
					match[i*n*m+l*n+k]+=w;
					suml+=w;
				}
				remainL[k]=fmaxf(0.0f,remainL[k]-suml);
			}
			__syncthreads();
		}
	}
}
void approxmatchLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,float * match){
	approxmatch<<<32,512>>>(b,n,m,xyz1,xyz2,match);
}
__global__ void matchcost(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,const float * __restrict__ match,float * __restrict__ out){
	__shared__ float allsum[512];
	const int Block=256;
	__shared__ float buf[Block*2];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		float subsum=0;
		for (int k0=0;k0<m;k0+=Block){
			int endk=min(m,k0+Block);
			for (int k=threadIdx.x;k<(endk-k0)*2;k+=blockDim.x){
				buf[k]=xyz2[i*m*2+k0*2+k];
			}
			__syncthreads();
			for (int j=threadIdx.x;j<n;j+=blockDim.x){
				float x1=xyz1[(i*n+j)*2+0];
				float y1=xyz1[(i*n+j)*2+1];
				for (int k=0;k<endk-k0;k++){
					//float x2=xyz2[(i*m+k)*2+0]-x1;
					//float y2=xyz2[(i*m+k)*2+1]-y1;
					float x2=buf[k*2+0]-x1;
					float y2=buf[k*2+1]-y1;
					float d=sqrtf(x2*x2+y2*y2);
					subsum+=match[i*n*m+(k0+k)*n+j]*d;
				}
			}
			__syncthreads();
		}
		allsum[threadIdx.x]=subsum;
		for (int j=1;j<blockDim.x;j<<=1){
			__syncthreads();
			if ((threadIdx.x&j)==0 && threadIdx.x+j<blockDim.x){
				allsum[threadIdx.x]+=allsum[threadIdx.x+j];
			}
		}
		if (threadIdx.x==0)
			out[i]=allsum[0];
		__syncthreads();
	}
}
void matchcostLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,const float * match,float * out){
	matchcost<<<32,512>>>(b,n,m,xyz1,xyz2,match,out);
}
__global__ void matchcostgrad(int b,int n,int m,const float * __restrict__ xyz1,const float * __restrict__ xyz2,const float * __restrict__ match,float * grad2){
	__shared__ float sum_grad[256*2];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		int kbeg=m*blockIdx.y/gridDim.y;
		int kend=m*(blockIdx.y+1)/gridDim.y;
		for (int k=kbeg;k<kend;k++){
			float x2=xyz2[(i*m+k)*2+0];
			float y2=xyz2[(i*m+k)*2+1];
			float subsumx=0,subsumy=0,subsumz=0;
			for (int j=threadIdx.x;j<n;j+=blockDim.x){
				float x1=x2-xyz1[(i*n+j)*2+0];
				float y1=y2-xyz1[(i*n+j)*2+1];
				float d=match[i*n*m+k*n+j]/fmaxf(sqrtf(x1*x1+y1*y1),1e-20f);
				subsumx+=x1*d;
				subsumy+=y1*d;
			}
			sum_grad[threadIdx.x*2+0]=subsumx;
			sum_grad[threadIdx.x*2+1]=subsumy;
			for (int j=1;j<blockDim.x;j<<=1){
				__syncthreads();
				int j1=threadIdx.x;
				int j2=threadIdx.x+j;
				if ((j1&j)==0 && j2<blockDim.x){
					sum_grad[j1*2+0]+=sum_grad[j2*2+0];
					sum_grad[j1*2+1]+=sum_grad[j2*2+1];
				}
			}
			if (threadIdx.x==0){
				grad2[(i*m+k)*2+0]=sum_grad[0];
				grad2[(i*m+k)*2+1]=sum_grad[1];
			}
			__syncthreads();
		}
	}
}
void matchcostgradLauncher(int b,int n,int m,const float * xyz1,const float * xyz2,const float * match,float * grad2){
	matchcostgrad<<<dim3(32,32),256>>>(b,n,m,xyz1,xyz2,match,grad2);
}

